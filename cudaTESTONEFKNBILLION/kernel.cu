#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string>
#define epsilon 0.000001

using namespace std;

void fillArrays(int* data, int* data2, int* backup, int size);
void copyArray(int* data, int* backup, int size);

void unoptimizedSort(int* randomNumbers, int size, FILE* file);
void testIfSorted(int* randomNumbers);
bool gpuSortingTest(int* data);

void cudaSort(int* &data, int size, int blocks, int tasksPerThread, FILE* file);
__global__ void oddEvenKernel(int* data, int size, int tasksPerThread, int index);

int main()
{
	srand(time(NULL));

	hipError_t cudaStatus = hipSuccess;
	FILE* file = fopen("data.txt", "w+");

	int* data, *data2, *backup;

	fprintf(file, "ODD-EVEN SORTING DATA\n---------------------------------------------\n");
	// Sorting, size 100, 1000, 10000, 100000
	for (int size = 100; size < 100001; size *= 10)
	{
		std::cout << "Working on size: " << size << std::endl;

		// Allocate memory for arrays
		data = (int*)malloc((size + 1) * sizeof(int));
		backup = (int*)malloc((size + 1) * sizeof(int));
		data2 = (int*)malloc((size + 1) * sizeof(int));

		// Fill arrays
		fillArrays(data, data2, backup, size);
		
		// CPU SORTING
		unoptimizedSort(data, size, file);

		// GPU SORTING
		for (int tasksPerThread = 1; tasksPerThread < 9; tasksPerThread *= 2)
		{
			std::cout << "Tasks per thread: " << tasksPerThread << std::endl;

			int threads = (size + 1) / tasksPerThread;
			int blocks = (threads - 1) / 1024 + 1; // 1024 to match current GPU limitations

			// Call GPU helper function
			cudaSort(data2, size, blocks, tasksPerThread, file);
		}
		std::cout << std::endl << "------------------------------------------------------------------" << std::endl;

		// Release array memory
		free(data);
		free(data2);
		free(backup);
	}
	

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset function in main failed.");
		return 1;
	}

	fclose(file);
	std::cout << "FINISHED! Press any key to exit." << std::endl;
	std::cin.get();

	return 0;
}


void fillArrays(int* data, int* data2, int* backup, int size)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = data2[i] = backup[i] = rand() % size + 1;
	}
}

void copyArray(int* data, int* backup, int size)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = backup[i];
	}
}

void unoptimizedSort(int* randomNumbers, int size, FILE* file)
{
	clock_t t;
	t = clock();

	bool sorted = false;

	// Loop until sorted
	while (!sorted)
	{
		int index = 0;
		sorted = true;

		// Sort even indices
		for (index; index < size - 2; index += 2)
		{
			if (randomNumbers[index] > randomNumbers[index + 1])
			{
				int temp = randomNumbers[index];
				randomNumbers[index] = randomNumbers[index + 1];
				randomNumbers[index + 1] = temp;
				sorted = false;
			}
		}
		//std::cout << "CPU - Finished sorting even indices" << std::endl;
		// Sort odd indices
		index = 1;
		for (index; index < size - 2; index += 2)
		{
			if (randomNumbers[index] > randomNumbers[index + 1])
			{
				int temp = randomNumbers[index];
				randomNumbers[index] = randomNumbers[index + 1];
				randomNumbers[index + 1] = temp;
				sorted = false;
			}
		}
		//std::cout << "CPU - Finished sorting odd indices" << std::endl;
	}
	std::cout << "CPU - Finished Sorting" << std::endl;
	t = clock() - t;

	std::cout << "CPU Odd-Even Sorting took: " << t << " clicks and " << ((float)t)/CLOCKS_PER_SEC << " seconds." << std::endl;
	fprintf(file, "CPU: %i %.4f \n", size,((float)t) / CLOCKS_PER_SEC);
	
	testIfSorted(randomNumbers);
}

void testIfSorted(int* randomNumbers)
{
	// Loop through array and check if sorted
	bool sorted = true;
	for (int i = 1; i < sizeof(randomNumbers); ++i)
	{
		if (randomNumbers[i] < randomNumbers[i - 1])
			sorted = false;
	}
	if (sorted)
		cout << endl << "The array is sorted!" << endl;
	else
		cout << endl << "The array is not sorted..." << endl;
}

bool gpuSortingTest(int* data)
{
	// Loop through array and check if sorted
	bool sorted = true;
	for (int i = 1; i < sizeof(data); ++i)
	{
		if (data[i] < data[i - 1])
			sorted = false;
	}
	return sorted;
}

// CUDA allocating function
void cudaSort(int* &data, int size, int blocks, int tasksPerThread, FILE* file)
{
	int* devArray = 0;
	clock_t t;
	t = clock();

	// Allocate array to GPU
	hipError_t cudaStatus = hipMalloc((void**)&devArray, (size + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for array\n");
		return;
	}
	// Copy array data to GPU
	cudaStatus = hipMemcpy(devArray, data, (size + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed for CPU -> GPU\n");
		return;
	}

	// Create temp array to retrieve array back from GPU
	int* tempArray = (int*)malloc((size + 1) * sizeof(int));

	// Call kernel function
	bool sorted = false;
	while (!sorted)
	{
		for (int i = 0; i < (size - 2); i += 2) // change how often its called
		{
			//cout << "Call GPU for even, current index: " << i << endl;
			oddEvenKernel << <blocks, 1024 >> > (devArray, size, tasksPerThread, i);
			//oddEvenKernel << <blocks, 1024 >> > (devArray, size, tasksPerThread, i * tasksPerThread);
		}
		for (int i = 1; i < (size - 2); i += 2) // change how often its called
		{
			oddEvenKernel << <blocks, 1024 >> > (devArray, size, tasksPerThread, i);
			//oddEvenKernel << <blocks, 1024 >> > (devArray, size, tasksPerThread, i * tasksPerThread);
		}

		// Retreive sorted array back from GPU
		cudaStatus = hipMemcpy((void*)tempArray, (void*)devArray, (size + 1) * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for GPU -> CPU\n");
			return;
		}
		sorted = gpuSortingTest(tempArray);
		//cout << "Sorted: " << sorted << endl;
	}


	data = tempArray;
	testIfSorted(data);

	t = clock() - t;
	std::cout << "GPU sorting took: " << t << "clicks (" << ((int)t) / CLOCKS_PER_SEC << " seconds.)" << endl;
	fprintf(file, "GPU %i: %.4i,", size, ((int)t) / CLOCKS_PER_SEC);

	hipFree(devArray);
	hipFree(tempArray);
}

// GPU Kernel function
__global__ void oddEvenKernel(int* data, int size, int tasksPerThread, int rowIndex)
{
	// Sort even indices
	if (data[rowIndex] > data[rowIndex + 1])
	{
		int temp = data[rowIndex];
		data[rowIndex] = data[rowIndex + 1];
		data[rowIndex + 1] = temp;
	}
}